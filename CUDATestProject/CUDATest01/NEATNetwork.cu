#include "Common.h"
#include "hip/hip_runtime.h"
#include ""




EXTERN_C{
	struct NEATLink
	{
		int toNeuron;
		int fromNeuron;
		float weight;
	};

	struct NEATNetworkParm
	{
		int linkCount;
		NEATLink *link;

		int neuronCount;
		int outputIndex;
		float *preActivation;
		float *postActivation;

		int inputCount;
		float *input;

		int outputCount;
		float *output;

		int activationCycles;
	};

	DLLTEST_API void __stdcall NEATNetwork_Work(NEATNetworkParm param)
	{
		// runTest(input, output, size);
	}

}

