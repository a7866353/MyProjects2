#include "hip/hip_runtime.h"
#include "Common.h"
#include "hip/hip_runtime.h"
#include ""

bool  getLevels(unsigned int len, unsigned int *levels);

__global__ void
Sumfloat_Kernel(const float *id, float *od, const int size)
{
	extern __shared__ float tmp[];
	for (int i = 1; i < blockDim.x; i <<= 1){
		if (threadIdx.x % (i << 1) == i){
			tmp[threadIdx.x - i] += tmp[threadIdx.x];
		}
		__syncthreads();
	}

	if (threadIdx.x == 0)    od[0] = tmp[0];
}




EXTERN_C{
	DLLTEST_API float __stdcall Sumfloat_CUDA(const float *inArr, int cntbuf)
	{
		float s = 0.0;
		float odata[1];
		float *d_idata = NULL;
		float *d_odata = NULL;
		unsigned int dlevels_left = 0;
		unsigned int dlevels_step = 0;

		getLevels(cntbuf, &dlevels_left);
		const unsigned int smem_size = sizeof(float)* cntbuf;
		hipMalloc((void **)&d_idata, smem_size);
		hipMalloc((void **)&d_odata, smem_size);
		// copy input data to device
		hipMemcpy(d_idata, inArr, smem_size, hipMemcpyHostToDevice);


		unsigned int num_threads_total_left = cntbuf / 2;

		dim3  block_size;
		dim3  grid_size;

		if (dlevels_left <= 10)
		{
			// decomposition can be performed at once
			block_size.x = num_threads_total_left;
		}
		else
		{
			// 512 threads per block
			grid_size.x = (num_threads_total_left / 512);
			block_size.x = 512;

			// 512 threads corresponds to 10 decomposition steps
			dlevels_step = 10;
			dlevels_left -= 10;

		}
		while (0 != num_threads_total_left)
		{
			// double the number of threads as bytes
			unsigned int mem_shared = (2 * block_size.x) * sizeof(float);

			// run kernel
			Sumfloat_Kernel << <grid_size, block_size, mem_shared >> >(d_idata, d_odata, cntbuf);
			// Copy approx_final to appropriate location

			// update level variables
			if (dlevels_left < 10)
			{
				// approx_final = d_odata;
			}

			// more global steps necessary
			dlevels_step = (dlevels_left > 10) ? dlevels_left - 10 : dlevels_left;
			dlevels_left -= 10;

			// after each step only half the threads are used any longer
			// therefore after 10 steps 2^10 less threads
			num_threads_total_left = num_threads_total_left >> 10;

			// update block and grid size
			grid_size.x = (num_threads_total_left / 512)
				+ (0 != (num_threads_total_left % 512)) ? 1 : 0;

			if (grid_size.x <= 1)
			{
				block_size.x = num_threads_total_left;
			}


		}
		hipMemcpy(odata, d_odata, sizeof(float)* 1,
			hipMemcpyDeviceToHost);

		return odata[0];
	}


}

