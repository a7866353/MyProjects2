#include "hip/hip_runtime.h"
#include "Common.h"
#include "hip/hip_runtime.h"
#include ""

bool  getLevels(unsigned int len, unsigned int *levels);

__global__ void
Sumfloat_Kernel(const float *id, float *od, const int size)
{
	extern __shared__ float tmp[];

	int idx = blockIdx.x * size * 2;
	tmp[threadIdx.x] = id[idx + threadIdx.x] + id[idx + threadIdx.x + blockDim.x];
	__syncthreads();

	for (int i = 1; i < blockDim.x; i <<= 1){
		if (threadIdx.x % (i << 1) == i){
			tmp[threadIdx.x - i] += tmp[threadIdx.x];
		}
		__syncthreads();
	}

	if (threadIdx.x == 0)    od[blockIdx.x] = tmp[0];
}

// #define D_NO_MALLOC
#ifdef D_NO_MALLOC
static float *gInData = NULL;
static float *gOutData = NULL;
#endif
EXTERN_C{

	void Sumfloat_CUDAMainCalc(float *d_idata, float *d_odata, int cntbuf)
	{
		unsigned int dlevels_left = 0;
		unsigned int dlevels_step = 0;

		getLevels(cntbuf, &dlevels_left);

		unsigned int num_threads_total_left = cntbuf / 2;

		dim3  block_size;
		dim3  grid_size;

		if (dlevels_left <= 10)
		{
			// decomposition can be performed at once
			block_size.x = num_threads_total_left;
		}
		else
		{
			// 512 threads per block
			grid_size.x = (num_threads_total_left / 512);
			block_size.x = 512;

			// 512 threads corresponds to 10 decomposition steps
			dlevels_step = 10;
			dlevels_left -= 10;

		}
		while (0 != num_threads_total_left)
		{
			// double the number of threads as bytes
			unsigned int mem_shared = (2 * block_size.x) * sizeof(float);

			// run kernel
			Sumfloat_Kernel << <grid_size, block_size, mem_shared >> >(d_idata, d_odata, block_size.x);
			// Copy approx_final to appropriate location

			// update level variables
			if (dlevels_left < 10)
			{
				// approx_final = d_odata;
			}

			// more global steps necessary
			dlevels_step = (dlevels_left > 10) ? dlevels_left - 10 : dlevels_left;
			dlevels_left -= 10;

			// after each step only half the threads are used any longer
			// therefore after 10 steps 2^10 less threads
			num_threads_total_left = num_threads_total_left >> 10;

			// update block and grid size
			grid_size.x = (num_threads_total_left / 512)
				+ (0 != (num_threads_total_left % 512)) ? 1 : 0;

			if (grid_size.x <= 1)
			{
				block_size.x = num_threads_total_left;
			}

			if (num_threads_total_left != 0)
			{
				float * temp = d_idata;
				d_idata = d_odata;
				d_odata = temp;
			}

		}
	}
	DLLTEST_API float __stdcall Sumfloat_CUDA(const float *inArr, int cntbuf)
	{
		float s = 0.0;
		float odata[1024*4*2];
		float *d_idata = NULL;
		float *d_odata = NULL;

		const unsigned int smem_size = sizeof(float)* cntbuf;
#ifndef D_NO_MALLOC
		hipMalloc((void **)&d_idata, smem_size);
		hipMalloc((void **)&d_odata, smem_size);
#else
		if (gInData == NULL)
		{
			hipMalloc((void **)&gInData, smem_size);
			hipMalloc((void **)&gOutData, smem_size);
		}
		d_idata = gInData;
		d_odata = gOutData;
#endif
		// copy input data to device
		hipMemcpy(d_idata, inArr, smem_size, hipMemcpyHostToDevice);

		Sumfloat_CUDAMainCalc(d_idata, d_odata, cntbuf);

		hipMemcpy(odata, d_odata, sizeof(float)* 1,
			hipMemcpyDeviceToHost);
#ifndef D_NO_MALLOC
		hipFree(d_idata);
		hipFree(d_odata);
#endif
		return odata[0];
	}

	DLLTEST_API void __stdcall CUDA_SumFloat(float *d_idata, float *d_odata, int cntbuf)
	{
		Sumfloat_CUDAMainCalc(d_idata, d_odata, cntbuf);
	}
	

	DLLTEST_API void* __stdcall CUDA_Malloc(int size)
	{
		void *addr;
		hipMalloc(&addr, size);
		return addr;
	}

	DLLTEST_API void __stdcall CUDA_Free(void *addr)
	{
		hipFree(addr);
	}

	DLLTEST_API void __stdcall CUDA_SetValue(void *cudaAddr, void *dataAddr, int size)
	{
		hipMemcpy(cudaAddr, dataAddr, size, hipMemcpyHostToDevice);
	}

	DLLTEST_API void __stdcall CUDA_GetValue(void *cudaAddr, void *dataAddr, int size)
	{
		hipMemcpy(dataAddr, cudaAddr, size, hipMemcpyDeviceToHost);

	}

}

